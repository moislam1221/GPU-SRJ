#include<utility>
#include<stdio.h>
#include<assert.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <ostream>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <fstream>
#include <omp.h>
#include <time.h>
#include <string.h>
#include <utility>
#include <time.h>
#include <iomanip>

// HEADER FILES
#include "Helper/fillThreadsPerBlock.h"
#include "Helper/level.h"
#include "Helper/jacobi.h"
#include "Helper/residual.h"
#include "Helper/setGPU.h"
#include "Helper/srjSchemes.h"

// #define RUN_CPU_FLAG 1
#define RUN_GPU_FLAG 1
// #define RUN_SHARED_FLAG 1

// Determine which header files to include based on which directives are active
#ifdef RUN_CPU_FLAG
#include "jacobi-1D-cpu.h"
#endif
#ifdef RUN_GPU_FLAG
#include "jacobi-1D-gpu.h"
#endif
#ifdef RUN_SHARED_FLAG
#include "jacobi-1D-shared-srj-shifted.h"
#endif

int main(int argc, char *argv[])
{
    /* Inputs and Settings */
    const int nDim = atoi(argv[1]); 
	const int threadsPerBlock = atoi(argv[2]);
	const int numCycles = atoi(argv[3]);
	const int levelSRJ = atoi(argv[4]);
    
    /* Set the correct GPU to use (Endeavour GPUs: "TITAN V" OR "GeForce GTX 1080 Ti") */
	std::string gpuToUse = "TITAN V"; // "GeForce GTX 1080 Ti"; 
    setGPU(gpuToUse);
    
	/* Initialize initial condition and rhs */
	const int Mcopies = 2;
    int nGrids = nDim + 2;
    double * initX = new double[nGrids * Mcopies];
    double * rhs = new double[nGrids * Mcopies];
    int stride;
	for (int Mcopy = 0; Mcopy < Mcopies; Mcopy++) {
		stride = Mcopy * nGrids;
		for (int iGrid = 0; iGrid < nGrids; ++iGrid) {
			if (iGrid == 0 || iGrid == nGrids-1) {
				initX[iGrid + stride] = 0.0f;
			}
			else {
				initX[iGrid + stride] = 1.0f;
			}
			rhs[iGrid + stride] = 1.0f;
		}
	}
    
	/* Load SRJ schemes from Python txt files */
	int numSchemes = 25;
	int numSchemeParams = 9710;
	double * srjSchemes = new double[numSchemeParams];
	int indexPointer[numSchemes] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
	loadSRJSchemes(srjSchemes, numSchemeParams);
	loadIndexPointer(indexPointer, numSchemes);

	/* CPU SRJ Jacobi*/
#ifdef RUN_CPU_FLAG
	printf("===============CPU SRJ============================\n");
    double cpuJacobiResidual;
    double * solutionJacobiCpu = new double[nGrids];
	// solutionJacobiCpu = jacobiCpuSRJ(initX, rhs, nGrids, srjSchemes, indexPointer, numSchemes, numCycles, levelSRJ);
	solutionJacobiCpu = jacobiCpuSRJHeuristic(initX, rhs, nGrids, srjSchemes, indexPointer, numSchemes, numCycles);
	cpuJacobiResidual = residual1DPoisson(solutionJacobiCpu, rhs, nGrids);
	printf("Residual of the Jacobi CPU solution is %f\n", cpuJacobiResidual);
#endif 

	/* GPU SRJ Jacobi */
#ifdef RUN_GPU_FLAG 
	printf("===============GPU SRJ============================\n");
    double gpuJacobiResidual;
    double * solutionJacobiGpu = new double[nGrids];
	float gpuSRJTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
 	solutionJacobiGpu = jacobiGpuSRJ(initX, rhs, nGrids, srjSchemes, indexPointer, numSchemes, threadsPerBlock, numCycles, levelSRJ, Mcopies);
 	// solutionJacobiGpu = jacobiGpuSRJHeuristic(initX, rhs, nGrids, srjSchemes, indexPointer, numSchemes, threadsPerBlock, numCycles);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuSRJTime, start, stop);
	gpuJacobiResidual = residual1DPoisson(solutionJacobiGpu, rhs, nGrids);
	printf("Residual of the Jacobi GPU solution is %.15f\n", gpuJacobiResidual);
	printf("Time needed for SRJ GPU: %f ms\n", gpuSRJTime);
/*	for (int i = 0; i < Mcopies * nGrids; i++) {
		printf("solutionJacobiGpu[%d] = %f\n", i, solutionJacobiGpu[i]);	
	}
*/
#endif 
	
	/* Shared SRJ Jacobi */
#ifdef RUN_SHARED_FLAG
	printf("===============SHARED SRJ============================\n");
    hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
    double sharedJacobiResidual;
    double * solutionJacobiShared = new double[nGrids];
	int overlap = 0;
	float sharedSRJTime;
	hipEvent_t start_shared, stop_shared;
	hipEventCreate(&start_shared);
	hipEventCreate(&stop_shared);
	hipEventRecord(start_shared, 0);	
	solutionJacobiShared = jacobiSharedSRJShifted(initX, rhs, nGrids, srjSchemes, indexPointer, numSchemes, numSchemeParams, threadsPerBlock, overlap, numCycles, levelSRJ, Mcopies);
	hipEventRecord(stop_shared, 0);	
	hipEventSynchronize(stop_shared);
	hipEventElapsedTime(&sharedSRJTime, start_shared, stop_shared);
	sharedJacobiResidual = residual1DPoisson(solutionJacobiShared, rhs, nGrids);
	printf("Residual of the Jacobi Shared solution is %.15f\n", sharedJacobiResidual);
	printf("Time needed for SRJ Shared: %f ms\n", sharedSRJTime);
/*	for (int i = 0; i < Mcopies * nGrids; i++) {
		printf("solutionJacobiShared[%d] = %f\n", i, solutionJacobiShared[i]);	
	}
*/
#endif 
   
    // FREE MEMORY
    delete[] initX;
    delete[] rhs;
	delete[] srjSchemes;
#ifdef RUN_CPU_FLAG
    delete[] solutionJacobiCpu;
#endif 
#ifdef RUN_GPU_FLAG 
    delete[] solutionJacobiGpu;
#endif
#ifdef RUN_SHARED_FLAG
    delete[] solutionJacobiShared;
#endif

    return 0;
}
