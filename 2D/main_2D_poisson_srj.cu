#include<utility>
#include<stdio.h>
#include<assert.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <ostream>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <fstream>
#include <omp.h>
#include <time.h>
#include <string.h>
#include <utility>
#include <time.h>

// HEADER FILES
#include "Helper/jacobi.h"
#include "Helper/residual.h"
#include "Helper/setGPU.h"
#include "Helper/solution_error.h"
#include "Helper/srjSchemes.h"

#define RUN_CPU_FLAG 1
#define RUN_GPU_FLAG 1
#define RUN_SHARED_FLAG 1

// Determine which header files to include based on which directives are active
#ifdef RUN_CPU_FLAG
#include "jacobi-2D-cpu.h"
#endif
#ifdef RUN_GPU_FLAG
#include "jacobi-2D-gpu.h"
#endif
#ifdef RUN_SHARED_FLAG
#include "jacobi-2D-shared-srj.h"
#endif

int main(int argc, char *argv[])
{
    /* Inputs and Settings */
    const int nxDim = atoi(argv[1]);
    const int nyDim = atoi(argv[1]); 
    const int threadsPerBlock_x = atoi(argv[2]); 
    const int threadsPerBlock_y = atoi(argv[2]); 
	const int numCycles = atoi(argv[3]);
	const int levelSRJ = atoi(argv[4]);
   
	/* Set the correct GPU to use (Endeavour GPUs: "TITAN V" OR "GeForce GTX 1080 Ti") */ 
	std::string gpuToUse = "TITAN V"; // "TITAN V";
    setGPU(gpuToUse);

    /* Initialize initial condition and rhs */
    int dof;
    int nxGrids = nxDim + 2;
    int nyGrids = nyDim + 2;
    int nDofs = nxGrids * nyGrids;
    double * initX = new double[nDofs];
    double * rhs = new double[nDofs];
    for (int jGrid = 0; jGrid < nyGrids; ++jGrid) {
        for (int iGrid = 0; iGrid < nxGrids; ++iGrid) {
            dof = iGrid + jGrid * nxGrids;
			if (iGrid == 0 || iGrid == nxGrids-1 || jGrid == 0 || jGrid == nyGrids-1) {
				initX[dof] = 0.0f;
			}
			else {
				initX[dof] = 1.0f; 
			}
			rhs[dof] = 1.0f;
        }
    }
	
	/* Load SRJ schemes from Python txt files */
	int numSchemes = 25;
	int numSchemeParams = 9710;
	double * srjSchemes = new double[numSchemeParams];
	int indexPointer[numSchemes] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
	loadSRJSchemes(srjSchemes, numSchemeParams);
	loadIndexPointer(indexPointer, numSchemes);
    
    /* CPU SRJ Jacobi */
#ifdef RUN_CPU_FLAG
    double cpuJacobiResidual;
    double * solutionJacobiCpu;
	solutionJacobiCpu = jacobiCpuSRJ(initX, rhs, nxGrids, nyGrids, srjSchemes, indexPointer, numSchemes, numCycles, levelSRJ);
	cpuJacobiResidual = residual2DPoisson(solutionJacobiCpu, rhs, nxGrids, nyGrids); 
	printf("Residual of the Jacobi CPU solution is %f\n", cpuJacobiResidual);
#endif

    /* GPU SRJ Jacobi */
#ifdef RUN_GPU_FLAG
    double gpuJacobiResidual;
	double * solutionJacobiGpu;
	solutionJacobiGpu = jacobiGpuSRJ(initX, rhs, nxGrids, nyGrids, srjSchemes, indexPointer, numSchemes, numCycles, levelSRJ, threadsPerBlock_x, threadsPerBlock_y);
	gpuJacobiResidual = residual2DPoisson(solutionJacobiGpu, rhs, nxGrids, nyGrids); 
	printf("Residual of the Jacobi GPU solution is %f\n", gpuJacobiResidual);
#endif

    /* Shared SRJ Jacobi */
#ifdef RUN_SHARED_FLAG
	hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
    double sharedJacobiResidual;
    double * solutionJacobiShared;  
    solutionJacobiShared = jacobiSharedSRJ(initX, rhs, nxGrids, nyGrids, srjSchemes, indexPointer, numSchemes, numSchemeParams, numCycles, levelSRJ, threadsPerBlock_x, threadsPerBlock_y, 0, 0);
    sharedJacobiResidual = residual2DPoisson(solutionJacobiShared, rhs, nxGrids, nyGrids);
	printf("Residual of the Jacobi Shared solution is %f\n", sharedJacobiResidual);
#endif
   
	/* Free Memory */
    delete[] initX;
    delete[] rhs;
	delete[] srjSchemes;
#ifdef RUN_CPU_FLAG
    delete[] solutionJacobiCpu;
#endif 
#ifdef RUN_GPU_FLAG 
    delete[] solutionJacobiGpu;
#endif
#ifdef RUN_SHARED_FLAG 
    delete[] solutionJacobiShared;
#endif
    
    return 0;
}
